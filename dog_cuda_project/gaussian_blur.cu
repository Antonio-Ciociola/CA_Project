#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ float d_kernel[64];
__device__ int KSIZE;
__device__ int WIDTH;
__device__ int HEIGHT;
__device__ int THRESHOLD;

__global__ void blur_horizontal(const unsigned char *input, unsigned char *output, float *d_kernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= WIDTH || y >= HEIGHT)
        return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i)
    {
        int ix = min(max(x + i, 0), WIDTH - 1);
        sum += input[y * WIDTH + ix] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = (unsigned char)(sum);
}

__global__ void blur_vertical(const unsigned char *input, unsigned char *output, float *d_kernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= WIDTH || y >= HEIGHT)
        return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i)
    {
        int iy = min(max(y + i, 0), HEIGHT - 1);
        sum += input[iy * WIDTH + x] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = (unsigned char)(sum);
}

__global__ void sumScale(const unsigned char *input1, const unsigned char *input2, unsigned char *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    output[y * width + x] = input1[y * width + x] - input2[y * width + x];
    output[y * width + x] = 255 - 20 * output[y * width + x];
    output[y * width + x] = min(max(output[y * width + x], 0), 255);
    output[y * width + x] = output > THRESHOLD ? 255 : 0;
}

void gaussian_blur_cuda(const uint8_t *input, uint8_t *output, int width, int height, float *kernel1, float *kernel2, int ksize, int threshold)
{

    // hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel.data(), sizeof(float) * ksize);

    uint8_t *d_input, *d_temp, *d_output, *d_kernel1, *d_kernel2, *d_out1, *d_out2;
    size_t img_size = width * height;
    size_t kernel_size = ksize * ksize;

    KSIZE = kernelsSize;
    WIDTH = width;
    HEIGHT = height;
    THRESHOLD = threshold;

    hipMalloc(&d_input, img_size);
    hipMalloc(&d_temp, img_size);
    hipMalloc(&d_out1, img_size);
    hipMalloc(&d_out2, img_size);
    hipMalloc(&d_output, img_size);
    hipMalloc(&d_kernel1, sizeof(float) * kernel_size);
    hipMalloc(&d_kernel2, sizeof(float) * kernel_size);

    hipMemcpy(d_input, input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel1, kernel1, sizeof(float) * kernel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel2, kernel2, sizeof(float) * kernel_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel1);
    blur_vertical<<<grid, block>>>(d_temp, d_out1, d_kernel1);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel2);
    blur_vertical<<<grid, block>>>(d_temp, d_out2, d_kernel2);

    sumScale<<<grid, block>>>(d_out1, d_out2, d_output);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
    hipFree(d_kernel1);
    hipFree(d_kernel2);

    /*
    if(hipGetLastError() != hipSuccess)
        cerr << "CUDA Error: " << hipGetErrorString(hipGetLastError()) << endl;
    */
}
