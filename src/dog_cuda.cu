#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;

uint8_t *d_input, *d_temp, *d_output, *d_out1, *d_out2;
float *d_kernel1, *d_kernel2;

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const unsigned char *input, unsigned char *output, float *d_kernel){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int ix = clamp(x + i, 0, WIDTH - 1);
        sum += input[y * WIDTH + ix] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = (unsigned char)(sum);
}

__global__ void blur_vertical(const unsigned char *input, unsigned char *output, float *d_kernel){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum += input[iy * WIDTH + x] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = (unsigned char)(sum);
}

__global__ void sumScale(const unsigned char *input1, const unsigned char *input2, unsigned char *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    unsigned char val = clamp(255 - 20*(int(input2[y * WIDTH + x]) - int(input1[y * WIDTH + x])), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, float* kernel1, float* kernel2, int ksize, float threshold = -1){
    size_t img_size = width * height;
    int i_threshold = threshold >= 0? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));
    
    hipMalloc(&d_input, img_size);
    hipMalloc(&d_temp, img_size);
    hipMalloc(&d_out1, img_size);
    hipMalloc(&d_out2, img_size);
    hipMalloc(&d_output, img_size);
    hipMalloc(&d_kernel1, sizeof(float) * ksize);
    hipMalloc(&d_kernel2, sizeof(float) * ksize);

    hipMemcpy(d_kernel1, kernel1, sizeof(float) * ksize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel2, kernel2, sizeof(float) * ksize, hipMemcpyHostToDevice);
}

void computeDoG(const uint8_t* input, uint8_t* output, int height, int width, float* _3, float* _4, int _5, float _6 = -1, int _7 = -1){
    size_t img_size = width * height;
    hipMemcpy(d_input, input, img_size, hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((width + 32 - 1) / 32, (height + 32 - 1) / 32);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel1);
    blur_vertical<<<grid, block>>>(d_temp, d_out1, d_kernel1);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel2);
    blur_vertical<<<grid, block>>>(d_temp, d_out2, d_kernel2);

    sumScale<<<grid, block>>>(d_out1, d_out2, d_output);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    if(hipGetLastError() != hipSuccess)
        cerr << "CUDA Error: " << hipGetErrorString(hipGetLastError()) << endl;
}

void finalize(){
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_out1);
    hipFree(d_out2);
    hipFree(d_output);
    hipFree(d_kernel1);
    hipFree(d_kernel2);
}
