#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;

uint8_t *d_input, *d_output;
float *d_temp, *d_out1, *d_out2;
float *d_kernel1, *d_kernel2;

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const unsigned char *input, float *output, float *d_kernel){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int ix = clamp(x + i, 0, WIDTH - 1);
        sum += input[y * WIDTH + ix] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void blur_vertical(const float *input, float *output, float *d_kernel){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum += input[iy * WIDTH + x] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void sumScale(const float *input1, const float *input2, unsigned char *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    unsigned char val = clamp(255 - 20*(input2[y * WIDTH + x] - input1[y * WIDTH + x]), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, float* kernel1, float* kernel2, int ksize, float threshold = -1){
    size_t img_size = width * height;
    int i_threshold = threshold >= 0? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));
    
    hipMalloc(&d_input, img_size);
    hipMalloc(&d_temp, sizeof(float) * img_size);
    hipMalloc(&d_out1, sizeof(float) * img_size);
    hipMalloc(&d_out2, sizeof(float) * img_size);
    hipMalloc(&d_output, img_size);
    hipMalloc(&d_kernel1, sizeof(float) * ksize);
    hipMalloc(&d_kernel2, sizeof(float) * ksize);

    hipMemcpy(d_kernel1, kernel1, sizeof(float) * ksize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel2, kernel2, sizeof(float) * ksize, hipMemcpyHostToDevice);
}

void computeDoG(const uint8_t* input, uint8_t* output, int height, int width, int _ = -1){
    size_t img_size = width * height;
    hipMemcpy(d_input, input, img_size, hipMemcpyHostToDevice);


    int xth=32, yth=32;

    dim3 block(xth, yth);
    dim3 grid((width + xth - 1) / xth, (height + yth - 1) / yth);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel1);
    blur_vertical<<<grid, block>>>(d_temp, d_out1, d_kernel1);

    blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel2);
    blur_vertical<<<grid, block>>>(d_temp, d_out2, d_kernel2);

    sumScale<<<grid, block>>>(d_out1, d_out2, d_output);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}

void finalize(){
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_out1);
    hipFree(d_out2);
    hipFree(d_output);
    hipFree(d_kernel1);
    hipFree(d_kernel2);
}
