#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

/*
0.023561        0.002902        0.002613        0.006297        0.035373
0.002119        0.000678        0.001876        0.004697        0.009370
0.001750        0.000667        0.002116        0.003860        0.008393
Read    Grayscale       DoG     Writer  Total
0.027430        0.004247        0.006605        0.014854        0.053136
*/

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;
__constant__ float c_kernel1[32], c_kernel2[32];

uint8_t *d_input, *d_output;
float *d_temp, *d_out1, *d_out2;

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const unsigned char *input, float *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    __half sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int ix = clamp(x + i, 0, WIDTH - 1);
        sum += input[y * WIDTH + ix] * c_kernel1[i + half];
    }

    output[y * WIDTH + x] = sum;
}
__global__ void blur_horizontal2(const unsigned char *input, float *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    __half sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int ix = clamp(x + i, 0, WIDTH - 1);
        sum += input[y * WIDTH + ix] * c_kernel2[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void blur_vertical(const float *input, float *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum += input[iy * WIDTH + x] * c_kernel1[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void blur_vertical2(const float *input, float *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    float sum = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i){
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum += input[iy * WIDTH + x] * c_kernel2[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void sumScale(const float *input1, const float *input2, unsigned char *output){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    unsigned char val = clamp(255 - 20*(input2[y * WIDTH + x] - input1[y * WIDTH + x]), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, float* kernel1, float* kernel2, int ksize, float threshold = -1, int _ = 1){
    size_t img_size = width * height;
    int i_threshold = threshold >= 0? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel1), kernel1, sizeof(float) * ksize);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kernel2), kernel2, sizeof(float) * ksize);
    
    hipMalloc(&d_input, img_size);
    hipMalloc(&d_temp, sizeof(float) * img_size);
    hipMalloc(&d_out1, sizeof(float) * img_size);
    hipMalloc(&d_out2, sizeof(float) * img_size);
    hipMalloc(&d_output, img_size);
}

void computeDoG(const uint8_t* input, uint8_t* output, int height, int width, int _ = -1, int _2 = 1, int xBlock = 32, int yBlock = 2) {
    size_t img_size = width * height;
    hipMemcpy(d_input, input, img_size, hipMemcpyHostToDevice);


    //const int xBlock = 32, yBlock = 4;

    dim3 block(xBlock, yBlock);
    dim3 grid((width + xBlock - 1) / xBlock, (height + yBlock - 1) / yBlock);

    blur_horizontal<<<grid, block>>>(d_input, d_temp);
    blur_vertical<<<grid, block>>>(d_temp, d_out1);

    blur_horizontal2<<<grid, block>>>(d_input, d_temp);
    blur_vertical2<<<grid, block>>>(d_temp, d_out2);

    sumScale<<<grid, block>>>(d_out1, d_out2, d_output);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}

void finalize(){
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_out1);
    hipFree(d_out2);
    hipFree(d_output);
}
