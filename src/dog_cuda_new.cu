#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;

__constant__ int tile_width;
__constant__ int tile_height;

uint8_t *d_input, *d_output;
float *d_temp, *d_out1, *d_out2;
float *d_kernel1, *d_kernel2;

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const unsigned char *input, float *output, float *d_kernel)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    extern __shared__ unsigned char tile[];

    int half = KSIZE / 2;
    unsigned char *tile_p = &tile[(ty + half) * tile_width + tx + half];

    int left = clamp(x - half, 0, WIDTH - 1);
    int right = clamp(x + half, 0, WIDTH - 1);

    tile_p[0] = input[y * WIDTH + x];
    tile_p[+half] = input[y * WIDTH + right];
    tile_p[-half] = input[y * WIDTH + left];
    __syncthreads();

    float sum = 0.0f;

    for (int i = -half; i <= half; ++i)
    {
        sum += tile_p[i] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void blur_vertical(const float *input, float *output, float *d_kernel)
{

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    extern __shared__ float f_tile[];

    int half = KSIZE / 2;

    float *tile_p = &f_tile[(ty + half) * tile_width + tx + half];

    int left = clamp(y - half, 0, HEIGHT - 1);
    int right = clamp(y + half, 0, HEIGHT - 1);

    tile_p[0] = input[y * WIDTH + x];
    tile_p[+half * tile_width] = input[right * WIDTH + x];
    tile_p[-half * tile_width] = input[left * WIDTH + x];

    __syncthreads();

    float sum = 0.0f;

    for (int i = -half; i <= half; ++i)
    {
        sum += tile_p[i * tile_width] * d_kernel[i + half];
    }

    output[y * WIDTH + x] = sum;
}

__global__ void sumScale(const float *input1, const float *input2, unsigned char *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    unsigned char val = clamp(255 - 20 * (input2[y * WIDTH + x] - input1[y * WIDTH + x]), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0 ? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, float *kernel1, float *kernel2, int ksize, float threshold = -1)
{
    size_t img_size = width * height;
    int i_threshold = threshold >= 0 ? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));

    hipMalloc(&d_input, img_size);
    hipMalloc(&d_temp, sizeof(float) * img_size);
    hipMalloc(&d_out1, sizeof(float) * img_size);
    hipMalloc(&d_out2, sizeof(float) * img_size);
    hipMalloc(&d_output, img_size);
    hipMalloc(&d_kernel1, sizeof(float) * ksize);
    hipMalloc(&d_kernel2, sizeof(float) * ksize);

    hipMemcpy(d_kernel1, kernel1, sizeof(float) * ksize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel2, kernel2, sizeof(float) * ksize, hipMemcpyHostToDevice);
}

void computeDoG(const uint8_t *input, uint8_t *output, int height, int width, int _ = -1)
{
    size_t img_size = width * height;
    hipMemcpy(d_input, input, img_size, hipMemcpyHostToDevice);

    const int xBlock = 32, yBlock = 32;

    dim3 block(xBlock, yBlock);
    dim3 grid((width + 32 - 1) / 32, (height + 32 - 1) / 32);

    int sharedWidth = block.x + KSIZE + 30;
    int sharedHeight = block.y + KSIZE + 30;
    int sharedMemSize = sharedWidth * sharedHeight *  (sizeof(unsigned char) + sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(tile_width), &sharedWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(tile_height), &sharedHeight, sizeof(int));

    blur_horizontal<<<grid, block, sharedMemSize>>>(d_input, d_temp, d_kernel1);
    blur_vertical<<<grid, block, sharedMemSize>>>(d_temp, d_out1, d_kernel1);

    blur_horizontal<<<grid, block, sharedMemSize>>>(d_input, d_temp, d_kernel2);
    blur_vertical<<<grid, block, sharedMemSize>>>(d_temp, d_out2, d_kernel2);

    sumScale<<<grid, block>>>(d_out1, d_out2, d_output);

    hipMemcpy(output, d_output, img_size, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}

void finalize()
{
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_out1);
    hipFree(d_out2);
    hipFree(d_output);
    hipFree(d_kernel1);
    hipFree(d_kernel2);
}
