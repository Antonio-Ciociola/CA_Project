#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;

uint8_t *d_input, *d_output;
float2 *d_temp;
__constant__ float2 d_kernels[16];

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const uint8_t *input, float2 *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    float2 sum;
    sum.x = 0.0f;
    sum.y = 0.0f;
    int half = KSIZE / 2;

    __shared__ uint8_t tile[4][80];
    int tx = threadIdx.x;
    int ty = threadIdx.y; // se metti più di 4 esplode fortissimo, sarà divertente

    // load itself (tile is offset by half to avoid negative indices)
    tile[ty][tx + half] = input[y * WIDTH + x];
    // if close to left edge, load left neighbor
    if(tx < half)
        tile[ty][tx] = input[y * WIDTH + (x - half < 0? 0 : x - half)];
    // if close to right edge, load right neighbor
    if(tx >= blockDim.x - half)
        tile[ty][tx + 2 * half] = input[y * WIDTH + (x + half >= WIDTH? WIDTH - 1 : x + half)];

    __syncthreads();

    #pragma unroll
    for(int i = 0; i < KSIZE; ++i){
        sum.x += tile[ty][tx + i] * d_kernels[i].x;
        sum.y += tile[ty][tx + i] * d_kernels[i].y;
    }

    output[y * WIDTH + x].x = sum.x;
    output[y * WIDTH + x].y = sum.y;
}

__global__ void blur_vertical(const float2 *input, uint8_t *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    float2 sum;
    sum.x = 0.0f;
    sum.y = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i)
    {
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum.x += input[iy * WIDTH + x].x * d_kernels[i + half].x;
        sum.y += input[iy * WIDTH + x].y * d_kernels[i + half].y;
    }

    uint8_t val = clamp(255 - 20 * (sum.y - sum.x), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0 ? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, float *kernel1, float *kernel2, int ksize, float threshold = -1, int batchSize = 1)
{
    size_t img_size = width * height;
    int i_threshold = threshold >= 0 ? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));

    hipMalloc(&d_input, sizeof(uint8_t) * img_size * batchSize);
    hipMalloc(&d_temp, sizeof(float2) * img_size);
    hipMalloc(&d_output, sizeof(uint8_t) * img_size * batchSize);

    float2 kernels[ksize];
    for(int i = 0; i < ksize; ++i){
        kernels[i].x = kernel1[i];
        kernels[i].y = kernel2[i];
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernels), kernels, sizeof(float2) * ksize);
}

void computeDoG(const uint8_t *input, uint8_t *output, int height, int width, int _ = -1, int batchSize = 1, int xBlock = 32, int yBlock = 2)
{
    size_t img_size = width * height;

    dim3 block(xBlock, yBlock);
    dim3 grid((width + xBlock - 1) / xBlock, (height + yBlock - 1) / yBlock);

    hipMemcpy(d_input, input, img_size * batchSize, hipMemcpyHostToDevice);

    for (int i = 0; i < batchSize; ++i){
        const uint8_t *batch_input = d_input + i * img_size; // Access the correct batch
        uint8_t *batch_output = d_output + i * img_size;     // Access the correct batch

        blur_horizontal<<<grid, block>>>(batch_input, d_temp);
        blur_vertical<<<grid, block>>>(d_temp, batch_output);
    }
    hipMemcpy(output, d_output, img_size * batchSize, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}

void finalize()
{
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
}
