#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <iostream>
using std::cerr;
using std::endl;

__constant__ int KSIZE;
__constant__ int WIDTH;
__constant__ int HEIGHT;
__constant__ int THRESHOLD;

uint8_t *d_input, *d_output;
float2 *d_temp;
float *d_kernel1, *d_kernel2;

#define clamp(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))

__global__ void blur_horizontal(const uint8_t *input, float2 *output, float *d_kernel1, float *d_kernel2)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    float2 sum;
    sum.x = 0.0f;
    sum.y = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i)
    {
        int ix = clamp(x + i, 0, WIDTH - 1);
        sum.x += (float)input[y * WIDTH + ix] * d_kernel1[i + half];
        sum.y += (float)input[y * WIDTH + ix] * d_kernel2[i + half];
    }

    output[y * WIDTH + x].x = sum.x;
    output[y * WIDTH + x].y = sum.y;
}

__global__ void blur_vertical(const float2 *input, uint8_t *output, float *d_kernel1, float *d_kernel2)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    float2 sum;
    sum.x = 0.0f;
    sum.y = 0.0f;
    int half = KSIZE / 2;

    for (int i = -half; i <= half; ++i)
    {
        int iy = clamp(y + i, 0, HEIGHT - 1);
        sum.x += input[iy * WIDTH + x].x * d_kernel1[i + half];
        sum.y += input[iy * WIDTH + x].y * d_kernel2[i + half];
    }

    uint8_t val = clamp(255 - 20 * (sum.y - sum.x), 0, 255);
    output[y * WIDTH + x] = THRESHOLD < 0 ? val : (val > THRESHOLD ? 255 : 0);
}

void initialize(int height, int width, int batchSize, float *kernel1, float *kernel2, int ksize, float threshold = -1)
{
    size_t img_size = width * height;
    int i_threshold = threshold >= 0 ? int(threshold) : -1;

    hipMemcpyToSymbol(HIP_SYMBOL(KSIZE), &ksize, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(WIDTH), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(HEIGHT), &height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(THRESHOLD), &i_threshold, sizeof(int));

    hipMalloc(&d_input, sizeof(uint8_t) * img_size * batchSize);
    hipMalloc(&d_temp, sizeof(float2) * img_size);
    hipMalloc(&d_output, sizeof(uint8_t) * img_size * batchSize);
    hipMalloc(&d_kernel1, sizeof(float) * ksize);
    hipMalloc(&d_kernel2, sizeof(float) * ksize);

    hipMemcpy(d_kernel1, kernel1, sizeof(float) * ksize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel2, kernel2, sizeof(float) * ksize, hipMemcpyHostToDevice);
}

void computeDoG(const uint8_t *input, uint8_t *output, int batchSize, int height, int width, int _ = -1)
{
    size_t img_size = width * height;

    const int xBlock = 32, yBlock = 2;
    dim3 block(xBlock, yBlock);
    dim3 grid((width + xBlock - 1) / xBlock, (height + yBlock - 1) / yBlock);

    hipMemcpy(d_input, input, img_size * batchSize, hipMemcpyHostToDevice);

    for (int i = 0; i < batchSize; ++i)
    {

        const uint8_t *batch_input = d_input + i * img_size; // Access the correct batch
        uint8_t *batch_output = d_output + i * img_size;     // Access the correct batch

        blur_horizontal<<<grid, block>>>(batch_input, d_temp, d_kernel1, d_kernel2);
        blur_vertical<<<grid, block>>>(d_temp, batch_output, d_kernel1, d_kernel2);
    }
    hipMemcpy(output, d_output, img_size * batchSize, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
}

void finalize()
{
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
    hipFree(d_kernel1);
    hipFree(d_kernel2);
}
